#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCNumerics.cuh>

#include "THC/THC.h"

#include "batch_norm.h"

#include <hip/hip_runtime.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

static size_t round_up_to_multiple(size_t x, int multiple) {
  return ((x + multiple - 1) / multiple) * multiple;
}

// TODO: Stop manually allocating CUDA memory; allocate an ATen byte
// tensor instead.
struct Workspace {
  Workspace(size_t size) : size(size), data(NULL) {
    data = THCudaMalloc(at::globalContext().lazyInitCUDA(), size);
  }
  Workspace(const Workspace&) = delete;
  Workspace(Workspace&&) = default;
  Workspace& operator=(Workspace&&) = default;
  ~Workspace() {
    if (data) {
      THCudaFree(at::globalContext().lazyInitCUDA(), data);
    }
  }

  size_t size;
  void* data;
};

// Return {y}
at::Tensor nhwc_bn_fwd_train(
                       const at::Tensor& x,
                       const at::Tensor& scale,
                       const at::Tensor& bias,
                       const at::Tensor& running_mean,
                       const at::Tensor& running_inv_var,
                       const at::Tensor& minibatch_mean,
                       const at::Tensor& minibatch_inv_var,
                       const float momentum,
                       const float epsilon,
                       const bool fuse_relu,
                       void * my_data,
                       void * pair_data,
                       void * pair_data2,
                       const int bn_group,
                       const at::Tensor& magic_tensor,
                       const int max_cta_per_sm,
                       const int cta_launch_margin) {

  const int N = x.size(0);
  const int H = x.size(1);
  const int W = x.size(2);
  const int C = x.size(3);

  // generating new magic number and use that for sync
  int* magic = magic_tensor.data<int>();
  *magic = (*magic + 1) & 0xff;

  // Allocate output tensor
  at::Tensor y = at::empty({N, H, W, C}, x.options());

  // Create wrapper
  NhwcBatchNorm *bn = new NhwcBatchNorm();

  bn->setInputDescriptor(HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, N, C, H, W, bn_group);
  bn->setOutputDescriptor(HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, N, C, H, W);

  bn->setConstants(momentum, epsilon);

  // set pointers within the wrapper
  bn->setInputOutputPointers(x.data<at::Half>(),
                             nullptr,
                             y.data<at::Half>(),
                             nullptr);

  bn->setWeightPointers({scale.data<float>(), bias.data<float>()}, {nullptr, nullptr});
  bn->setParameterPointers({running_mean.data<float>(), running_inv_var.data<float>()});

  // deal with workspace(s)
  auto workspace_bytes = bn->numWorkspaceBytes();
  // We'll create explicit tensors for the first 2 workspace ptrs, then allocate & offset
  // an allocated workspace for the others
  size_t total_workspace_bytes = 0;
  std::vector<size_t> workspace_offsets;

  for (auto index = 3; index < workspace_bytes.size(); ++index) {
    total_workspace_bytes = round_up_to_multiple(total_workspace_bytes, 512);
    workspace_offsets.push_back(total_workspace_bytes);

    auto alloc_bytes = workspace_bytes[index];
    total_workspace_bytes += alloc_bytes;
  }

  // Allocate the workspace
  Workspace ws(total_workspace_bytes);

  std::vector<void *> workspace;
  workspace.push_back(minibatch_mean.data<float>());
  workspace.push_back(minibatch_inv_var.data<float>());

  auto stream = at::cuda::getCurrentCUDAStream().stream();
  const int retired_cta_bytes = workspace_bytes[2];
  void* retired_ctas = THCudaMalloc(at::globalContext().lazyInitCUDA(), retired_cta_bytes); 
  hipMemsetAsync(retired_ctas, 0, retired_cta_bytes, stream); //FIXME: is this legit?
  workspace.push_back(retired_ctas);

  for (auto index = 3; index < workspace_bytes.size(); ++index) {
    void *ptr = reinterpret_cast<uint8_t*>(ws.data) + workspace_offsets[index-3];
    workspace.push_back(ptr);
  }

  bn->setWorkspacePointers(workspace, workspace_bytes);

  int device_id;
  hipGetDevice(&device_id);
  // Don't fuse in ReLU for now at least
  bn->fwd(stream, fuse_relu, device_id, my_data, pair_data, pair_data2, bn_group, *magic, max_cta_per_sm, cta_launch_margin);

  THCudaFree(at::globalContext().lazyInitCUDA(), retired_ctas);
  return y;
}

at::Tensor nhwc_bn_fwd_eval(
                       const at::Tensor& x,
                       const at::Tensor& scale,
                       const at::Tensor& bias,
                       const at::Tensor& running_mean,
                       const at::Tensor& running_inv_var,
                       const int bn_group,
                       const float momentum,
                       const float epsilon,
                       const bool fuse_relu) {

  const int N = x.size(0);
  const int H = x.size(1);
  const int W = x.size(2);
  const int C = x.size(3);

  // Allocate output tensor
  at::Tensor y = at::empty({N, H, W, C}, x.options());

  // Create wrapper
  NhwcBatchNorm *bn = new NhwcBatchNorm();

  bn->setInputDescriptor(HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, N, C, H, W, bn_group);
  bn->setOutputDescriptor(HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, N, C, H, W);

  bn->setConstants(momentum, epsilon);

  // set pointers within the wrapper
  bn->setInputOutputPointers(x.data<at::Half>(),
                             nullptr,
                             y.data<at::Half>(),
                             nullptr);

  bn->setWeightPointers({scale.data<float>(), bias.data<float>()}, {nullptr, nullptr});
  bn->setParameterPointers({running_mean.data<float>(), running_inv_var.data<float>()});

  // deal with workspace(s)
  auto workspace_bytes = bn->numWorkspaceBytes();
  // We'll create explicit tensors for the first 2 workspace ptrs, then allocate & offset
  // an allocated workspace for the others
  size_t total_workspace_bytes = 0;
  std::vector<size_t> workspace_offsets;

  for (auto index = 3; index < workspace_bytes.size(); ++index) {
    total_workspace_bytes = round_up_to_multiple(total_workspace_bytes, 512);
    workspace_offsets.push_back(total_workspace_bytes);

    auto alloc_bytes = workspace_bytes[index];
    total_workspace_bytes += alloc_bytes;
  }

  // Allocate the workspace
  Workspace ws(total_workspace_bytes);

  std::vector<void *> workspace;
  workspace.push_back(nullptr);
  workspace.push_back(nullptr);

  auto stream = at::cuda::getCurrentCUDAStream().stream();
  const int retired_cta_bytes = workspace_bytes[2];
  void* retired_ctas = THCudaMalloc(at::globalContext().lazyInitCUDA(), retired_cta_bytes);
  hipMemsetAsync(retired_ctas, 0, retired_cta_bytes, stream); //FIXME: is this legit?
  workspace.push_back(retired_ctas);

  for (auto index = 3; index < workspace_bytes.size(); ++index) {
    void *ptr = reinterpret_cast<uint8_t*>(ws.data) + workspace_offsets[index-3];
    workspace.push_back(ptr);
  }

  bn->setWorkspacePointers(workspace, workspace_bytes);

  // Don't fuse in ReLU for now at least
  bn->fwdInference(stream, fuse_relu);

  THCudaFree(at::globalContext().lazyInitCUDA(), retired_ctas);
  return y;

}

std::vector<at::Tensor> nhwc_bn_bwd(
                       const at::Tensor& x,
                       const at::Tensor& dy,
                       const at::Tensor& scale,
                       const at::Tensor& bias,
                       const at::Tensor& running_mean,
                       const at::Tensor& running_inv_var,
                       const at::Tensor& minibatch_mean,
                       const at::Tensor& minibatch_inv_var,
                       const float momentum,
                       const float epsilon,
                       const bool fuse_relu,
                       void * my_data,
                       void * pair_data, 
                       void * pair_data2, 
                       const int bn_group,
                       const at::Tensor& magic_tensor,
                       const int max_cta_per_sm,
                       const int cta_launch_margin) {
  // shape
  const int N = x.size(0);
  const int H = x.size(1);
  const int W = x.size(2);
  const int C = x.size(3);

  // generating new magic number and use that for sync
  int* magic = magic_tensor.data<int>();
  *magic = (*magic + 1) & 0xff;

  // outputs
  at::Tensor x_grad, scale_grad, bias_grad;

  // Allocate outputs
  x_grad = at::empty_like(x);
  scale_grad = at::empty_like(scale);
  bias_grad = at::empty_like(bias);

  // Create wrapper
  NhwcBatchNorm *bn = new NhwcBatchNorm();

  bn->setInputDescriptor(HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, N, C, H, W, bn_group);
  bn->setOutputDescriptor(HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, N, C, H, W);

  bn->setConstants(momentum, epsilon);

  // set pointers within the wrapper
  bn->setInputOutputPointers(x.data<at::Half>(),
                             x_grad.data<at::Half>(),
                             nullptr,
                             dy.data<at::Half>());

  bn->setWeightPointers({scale.data<float>(), bias.data<float>()}, {scale_grad.data<float>(), bias_grad.data<float>()});
  bn->setParameterPointers({running_mean.data<float>(), running_inv_var.data<float>()});

  // deal with workspace(s)
  auto workspace_bytes = bn->numWorkspaceBytes();
  // We'll create explicit tensors for the first 2 workspace ptrs, then allocate & offset
  // an allocated workspace for the others
  size_t total_workspace_bytes = 0;
  std::vector<size_t> workspace_offsets;

  for (auto index = 3; index < workspace_bytes.size(); ++index) {
    total_workspace_bytes = round_up_to_multiple(total_workspace_bytes, 512);
    workspace_offsets.push_back(total_workspace_bytes);

    auto alloc_bytes = workspace_bytes[index];
    total_workspace_bytes += alloc_bytes;
  }

  // Allocate the workspace
  Workspace ws(total_workspace_bytes);

  std::vector<void *> workspace;
  workspace.push_back(minibatch_mean.data<float>());
  workspace.push_back(minibatch_inv_var.data<float>());

  auto stream = at::cuda::getCurrentCUDAStream().stream();
  const int retired_cta_bytes = workspace_bytes[2];
  void* retired_ctas = THCudaMalloc(at::globalContext().lazyInitCUDA(), retired_cta_bytes);
  hipMemsetAsync(retired_ctas, 0, retired_cta_bytes, stream); //FIXME: is this legit?
  workspace.push_back(retired_ctas);

  for (auto index = 3; index < workspace_bytes.size(); ++index) {
    void *ptr = reinterpret_cast<uint8_t*>(ws.data) + workspace_offsets[index-3];
    workspace.push_back(ptr);
  }

  bn->setWorkspacePointers(workspace, workspace_bytes);

  int device_id;
  hipGetDevice(&device_id);
  bn->dgrad(stream, fuse_relu, device_id, my_data, pair_data, pair_data2, bn_group, *magic, max_cta_per_sm, cta_launch_margin);

  THCudaFree(at::globalContext().lazyInitCUDA(), retired_ctas);
  return std::vector<at::Tensor>{x_grad, scale_grad, bias_grad};
}
